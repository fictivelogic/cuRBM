#include "hip/hip_runtime.h"
//
// Created by samuel on 5/24/15.
//

#include <hip/hip_runtime.h>
#include "createMovieRatings.h"


__global__ void
createMovieRatingsKernel(const float *weights, const float *hidden_features,
    float* movie_rating_probs, int num_movies, int num_hidden_features) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    //
    unsigned int movie_rating_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = 0;
    float dot_prod;
    while (movie_rating_index < num_movies * 5) {
        dot_prod = 0.00; // Initialize the dot product to 0

        for (i = 0; i < num_hiddens; i++) {
            // Indexing: weights[movie_id][rating_id][feature_id]
            // movie_id - [1, 17771]
            // rating_id - [0, 4]
            // feature_id - [0, 99]
            dot_prod += weights[movie_rating_index*num_hidden_features + i]
                        + hidden_features[i]; // Do the dot product
        }
        movie_rating_probs[movie_rating_index] = dot_prod;
        movie_rating_index += blockDim.x * gridDim.x;
    }
}
