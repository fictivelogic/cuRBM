#include "hip/hip_runtime.h"
#include <cassert>
#include <math.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"

#define NUM_MOVIES 17770
#define NUM_USERS 458293

__global__
void createInitialHidden(const int * const train_points,
						 const int * user_start,
						 const int * user_length,
						 int num_hidden,
						 float * output) {

  // TODO: do not modify code, just comment on suboptimal accesses

	int i_start = user_start(threadIdx.x + (blockDim.x * threadIdx.y));


	const int i = threadIdx.x + 64 * blockIdx.x;
	int j = 4 * threadIdx.y + 64 * blockIdx.y;
	const int end_j = j + 4;

	for (; j < end_j; j++) {
	  output[j + n * i] = input[i + n * j];
	}
}