#include "hip/hip_runtime.h"
#include <cassert>
#include <math.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"

#define NUM_MOVIES 17770
#define NUM_USERS 458293


float sigmoid (float input); //TODO: Log probability?

__global__
void createInitialHidden(const int * const train_points,
						 const int * user_start,
						 const int * user_length,
						 const int * const b_hid,
						 float * intial_hiddens,
						 int num_hidden,
						 float * output) {

  // TODO: do not modify code, just comment on suboptimal accesses
	//

	int i_start = user_start[threadIdx.x + (blockDim.x * threadIdx.y)];
	int i_length = user_length[threadIdx.x + (blockDim.x * threadIdx.y)];
	//Need to know what size the thread block is so we can determine how many users each thread should attend to

	for (int i = 0; i < num_hidden < i++) //100 loops
	{
		float dot_prod = 0;
		for (int m = 0; m < i_length; m++)
		{
			int rating = train_points[i_start + (m * 2)];
			dot_prod += W[movie][rating][i];
			intial_hiddens[i] = sigmoid(dot_prod - b_hid[i]);
		}
	}
}