#include "hip/hip_runtime.h"
//
// Created by samuel on 5/24/15.
//

#include <hip/hip_runtime.h>
#include "createMovieRatings.cuh"


__global__ void
createMovieRatingsKernel(const float *weights, const float *hidden_features,
    float* movie_rating_probs, int num_movies, int num_hidden_features) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    //
    // movie_rating_index = movie_id * 5 + rating_id
    //      (index of current movie_id/rating_id pair)
    unsigned int movie_rating_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = 0;
    float dot_prod; // temporary, local dot product variable
    while (movie_rating_index < num_movies * 5) {
        dot_prod = 0.00; // Initialize the dot product to 0

        for (i = 0; i < num_hiddens; i++) {
            // Indexing: weights[movie_id][rating_id][feature_id]
            // movie_id - [1, 17771]
            // rating_id - [0, 4]
            // feature_id - [0, 99]
            dot_prod += weights[movie_rating_index*num_hidden_features + i]
                        * hidden_features[i]; // Do the dot product
        }
        // store the dot_product result
        movie_rating_probs[movie_rating_index] = dot_prod;

        // re-use this thread on another data point:
        movie_rating_index += blockDim.x * gridDim.x;
    }
}
