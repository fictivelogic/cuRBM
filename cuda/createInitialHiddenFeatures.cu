#include "hip/hip_runtime.h"
__global__ void
createInitialHiddenFeaturesKernel(const float *weights,
    const int *movie_ratings, float* initial_hidden_feature_probs,
    int num_movies, int num_hidden_features, int num_user_ratings) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    // movie_ratings[NUM_TRAIN_POINTS][3]
    // initial_hidden_feature_probs[NUM_FEATURES]
    unsigned int hidden_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int point_id = 0;
    float dot_prod; // Temporary, local dot product variable
    while (hidden_id < num_hidden_features) {
        dot_prod = 0.00; // Initialize the dot product to 0

        for (point_id = 0; point_id < num_user_ratings; point_id++) {
            // Indexing: weights[movie_id][rating][feature_id]
            // user_id - [1, 500,000]
            // movie_id - [1, 17771]
            // rating - [0, 4]
            // hidden_id - [0, 99]
            user_id = *movie_ratings++;
            movie_id = *movie_ratings++;
            rating = *movie_ratings++;
            // Do the dot product
            dot_prod += weights[movie_id*5*num_hidden_features
					            + rating*num_hidden_features
					            + hidden_id]
                        * initial_hidden_feature_probs[hidden_id];
        }
        // Store the dot_product result
        initial_hidden_feature_probs[hidden_id] = dot_prod;

        // Re-use this thread on another data point:
        hidden_id += blockDim.x * gridDim.x;
    }
}
