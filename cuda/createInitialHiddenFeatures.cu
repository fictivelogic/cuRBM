#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "createInitialHiddenFeatures.cuh"

__global__ void
createInitialHiddenFeaturesKernel(const float *weights,
    const int *movie_ratings, float* initial_hidden_feature_probs,
    int num_movies, int num_hidden_features) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    // movie_ratings[NUM_MOVIES][5]
    // initial_hidden_feature_probs[NUM_FEATURES]
}
