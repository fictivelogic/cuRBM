#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "createInitialHiddenFeatures.cuh"

__global__ void
createInitialHiddenFeaturesKernel(const float *weights,
    const int *movie_ratings, float* initial_hidden_feature_probs,
    int num_movies, int num_hidden_features, int num_user_ratings) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    // movie_ratings[NUM_TRAIN_POINTS][3]
    // initial_hidden_feature_probs[NUM_FEATURES]
    unsigned int hidden_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int point_index = 0;
    float dot_prod; // Temporary, local dot product variable
    while (hidden_index < num_hidden_features) {
        dot_prod = 0.00; // Initialize the dot product to 0

        for (point_index = 0; point_index < num_user_ratings; point_index++) {
            // Indexing: weights[movie_id][rating_id][feature_id]
            // movie_id - [1, 500,000]
            // movie_id - [1, 17771]
            // rating_id - [0, 4]
            // hidden_index - [0, 99]
            user_id = *movie_ratings++;
            movie_id = *movie_ratings++;
            rating = *movie_ratings++;
            // Do the dot product
            dot_prod += weights[movie_id*5*num_hidden_features
					            + rating*num_hidden_features
					            + hidden_index]
                        * initial_hidden_feature_probs[];
        }
        // Store the dot_product result
        movie_rating_probs[movie_rating_index] = dot_prod;

        // Re-use this thread on another data point:
        hidden_index += blockDim.x * gridDim.x;
    }
}
