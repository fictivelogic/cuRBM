#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "createFinalHiddenFeatures.cuh"

__global__ void
createFinalHiddenFeaturesKernel(const float *weights,
    const float *movie_rating_probs, float* final_hidden_feature_probs,
    int num_movies, int num_hidden_features) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    // movie_rating_probs[NUM_MOVIES][5]
    // final_hidden_feature_probs[NUM_FEATURES]
}
